#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>

#include <cub/block/block_load.cuh>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

#include "includes/block_reduce.h"
#include "includes/kernels.h"

#include <hip/hip_cooperative_groups.h>

namespace flash_attn{
namespace cuda{

// remember that br/bc might divide seq_len

template <typename T>
__global__ void flash_attn_fw(const T *Q, const T* K, const T* V, T* O, T* L, T* M, int seq_len, int head_dim,const T * masks, bool is_causal) {
    int batch_id = blockIdx.y;
    int head_id = blockIdx.z;
    int batch_size = gridDim.y;
    int nhead = gridDim.z;
    int br = blockDim.y;
    int bc = blockDim.x;
    int outer_steps = (seq_len + bc - 1) / bc;
    int inner_steps = (seq_len + br - 1) / br;

    int stride_batch = nhead * seq_len * head_dim;
    int stride_head = seq_len * head_dim;
    int stride_seq = head_dim;
    // typedef hipcub::BlockLoad<T, block_dim, ele_per_thread,
    //                      hipcub::BLOCK_LOAD_VECTORIZE>
    //     BlockLoad;
    // __shared__ typename BlockLoad::TempStorage ts_load;
    // typedef hipcub::BlockStore<T, block_dim, ele_per_thread,
    //                         hipcub::BLOCK_STORE_VECTORIZE>
    //     BlockStore;
    // __shared__ typename BlockStore::TempStorage ts_store;

    __shared__ T shared_k[bc][head_dim];
    __shared__ T shared_v[bc][head_dim];
    __shared__ T shared_q[br][head_dim];
    __shared__ T shared_o[br][head_dim];
    __shared__ T shared_l[br];
    __shared__ T shared_l_ij[br];
    __shared__ T shared_l_new[br];
    __shared__ T shared_m[br];
    __shared__ T shared_m_ij[br];
    __shared__ T shared_m_new[br];
    __shared__ T shared_s[br][bc];
    
    // __shared__ T shared_q[br][head_dim];
    // __shared__ T shared_o[br][bc];
    for (int j=0;j<outer_steps;++j){
        // load KV to on-chip memory
        int kv_per_thread = (head_dim + br -1) / br;
        for (int col_idx = 0; col_idx < kv_per_thread; ++col_idx){
            int ele_idx = threadIdx.y * num_per_thread + col_idx;
            if (ele_idx < head_dim){
                shared_k[threadIdx.x][ele_idx] = K[batch_id * stride_batch + head_id * stride_head + (j * bc + threadIdx.x) * stride_seq + ele_idx];
                shared_v[threadIdx.x][ele_idx] = V[batch_id * stride_batch + head_id * stride_head + (j * bc + threadIdx.x) * stride_seq + ele_idx];
            }
        }
        __syncthreads();
        // inner loop
        for (int i = 0;i<inner_steps;++i){
            //load Q to on-chip memory
            int qo_per_thread = (head_dim + br -1) / br;
            for (int col_idx = 0; col_idx < qo_per_thread; ++col_idx){
                int ele_idx = threadIdx.x * num_per_thread + col_idx;
                if (ele_idx < head_dim){
                    shared_q[threadIdx.y][ele_idx] = Q[batch_id * stride_batch + head_id * stride_head + (i * br + threadIdx.y) * stride_seq + ele_idx];
                }
            }
            // always true for threadIdx.y < bc
            // load l and m to on-chip memory
            if (threadIdx.x == 0){
                shared_l[threadIdx.y] = L[batch_id * n_head * seq_len + head_id * seq_len + (i * br + threadIdx.y)]; 
                shared_m[threadIdx.y] = M[batch_id * n_head * seq_len + head_id * seq_len + (i * br + threadIdx.y)];
            }
            __syncthreads();
            // compute attention
            T sum_ = 0;
            for (int k = 0; k < head_dim; ++k){
                sum_ += shared_q[threadIdx.y][k] * shared_k[threadIdx.x][k];
            }
            shared_s[threadIdx.y][threadIdx.x] = sum_;
            
            __syncthreads();

            // row-wise computation for the normalization factors
            if (threadIdx.x == 0){
                T m_ij_ = 0;
                for (int k = 0; k < bc; ++k){
                    m_ij_ = max(m_ij_, shared_s[threadIdx.y][k]);
                }
                shared_m_ij[threadIdx.y] = mij_;
                // softmax
                T l_ij_ = 0;
                for (int k = 0; k < bc; ++k){
                    shared_s[threadIdx.y][k] = __expf(shared_s[threadIdx.y][k] - m_ij_);
                    l_ij_ += shared_s[threadIdx.y][k];
                }
                shared_l_ij[threadIdx.y] = l_ij_;

                // compute l and m
                T m_new_ = max(shared_m[threadIdx.y], m_ij_);
                T l_new_ = __expf(shared_m[threadIdx.y] - m_new_) * shared_l[threadIdx.y] + __expf(m_ij_ - m_new_) * l_ij_;
                
                // need to write back to shared_memory for row sharing
                shared_l_new[threadIdx.y] = l_new_;
                shared_m_new[threadIdx.y] = m_new_;
            }
            __syncthreads();

            /*
            Compute O on-chip and write back to HBM
            */
            // step 1: compute the shared_o
            T factor_o = shared_l[threadIdx.y] * __expf(shared_m[threadIdx.y] - shared_m_new[threadIdx.y]) / shared_l_new[threadIdx.y];
            T factor_pv = __expf(shared_m_ij[threadIdx.y] - shared_m_new[threadIdx.y]) / shared_l_new[threadIdx.y];
            for (int col_idx = 0;col_idx<qo_per_thread; ++col_idx){
                int ele_idx = threadIdx.x * num_per_thread + col_idx;
                if (ele_idx < head_dim){
                    T sum_pv = 0;
                    for (int k = 0; k<bc; ++k){
                        sum_pv += shared_s[threadIdx.y][k] * shared_v[k][ele_idx];
                    }
                    sum_pv *= factor_pv;
                    T sum_o = shared_o[threadIdx.y][ele_idx] * factor_o + sum_pv;
                    shared_o[threadIdx.y][ele_idx] = sum_o;
                }
            }
            __syncthreads();

            // step 2: write back to global O
            for (int col_idx = 0; col_idx < qo_per_thread; ++col_idx){
                int ele_idx = threadIdx.x * num_per_thread + col_idx;
                if (ele_idx < head_dim){
                    O[batch_id * stride_batch + head_id * stride_head * (i * br + threadIdx.y) * stride_seq + ele_idx] = shared_o[threadIdx.y][ele_idx];
                }
            }
            if (threadIdx.x==0){
                shared_l[threadIdx.y] = shared_l_new[threadIdx.y];
                shared_m[threadIdx.y] = shared_m_new[threadIdx.y];
            }
            __syncthreads();

        }

    }

}
    

extern "C" {
void launch_flash_attn_fw(const float *Q, const float* K, const float * V, float * O, 
                                int batch_size, int nhead, int seq_len, int head_dim,
                                bool is_causal,
                                hipStream_t stream) {

  int float_size = sizeof(float);
  int qkv_size = batch_size * nhead * seq_len * head_dim * float_size;

  float *d_q, *d_k, *d_v, *d_o;
  hipMalloc((void **)&d_q, qkv_size);
  hipMalloc((void **)&d_k, qkv_size);
  hipMalloc((void **)&d_v, qkv_size);
  hipMalloc((void **)&d_o, qkv_size);


  hipMemcpy(d_q, Q, qkv_size, hipMemcpyHostToDevice);
  hipMemcpy(d_k, K, qkv_size, hipMemcpyHostToDevice);
  hipMemcpy(d_v, V, qkv_size, hipMemcpyHostToDevice);


  dim3 grid_dim(1, batch_size, nhead);

  // get shared memory size M
  hipDeviceProp_t prop;
  int deviceId;
  hipGetDevice(&deviceId);
  hipGetDeviceProperties(&prop, deviceId);
  int M = prop.sharedMemPerBlock;

  // calculate block size
  int bc, br;
  bc = min(M/(4*head_dim), 32);
  br = min(M/(4*head_dim), 32);

  dim3 block_dim(bc, br);

  // launch kernel
  flash_attn_fw<float><<<grid_dim, block_dim, 0, stream>>>(d_q, d_k, d_v, d_o, head_dim, nullptr, is_causal);
  
  // Copy back to the host
  hipMemcpy(O, d_o, qkv_size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  // Check CUDA execution
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "launch_attn_softmax Error: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Free memory on device
  hipFree(d_q);
  hipFree(d_k);
  hipFree(d_v);
  hipFree(d_o);
  
}}


    
}
}