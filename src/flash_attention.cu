#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>

#include <cub/block/block_load.cuh>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

#include "includes/block_reduce.h"
#include "includes/kernels.h"

#include <hip/hip_cooperative_groups.h>

namespace flash_attn{
namespace cuda{

// called from global, use pointer to modify the bias in-place
template <typename T>
__device__ T* GetSharedPtr(T * shared_mem, int * ptr_bias, int mem_size) {
    T * cur_shared_mem = shared_mem + *ptr_bias;
    *ptr_bias += mem_size;
    return cur_shared_mem;
}
// remember that br/bc might divide seq_len


template <typename T>
__global__ void flash_attn_fw(const T *Q, const T* K, const T* V, T* O, T* L, T* M, T* attn_mask, int seq_len, int head_dim,const T * masks, bool is_causal) {
    int batch_id = blockIdx.y;
    int head_id = blockIdx.x;
    int batch_size = gridDim.y;
    int nhead = gridDim.x;
    int br = blockDim.y;
    int bc = blockDim.x;
    int outer_steps = (seq_len + bc - 1) / bc;
    int inner_steps = (seq_len + br - 1) / br;

    int stride_batch = nhead * seq_len * head_dim;
    int stride_head = seq_len * head_dim;
    int stride_seq = head_dim;
    // typedef hipcub::BlockLoad<T, block_dim, ele_per_thread,
    //                      hipcub::BLOCK_LOAD_VECTORIZE>
    //     BlockLoad;
    // __shared__ typename BlockLoad::TempStorage ts_load;
    // typedef hipcub::BlockStore<T, block_dim, ele_per_thread,
    //                         hipcub::BLOCK_STORE_VECTORIZE>
    //     BlockStore;
    // __shared__ typename BlockStore::TempStorage ts_store;

    extern __shared__ T shared_mem[];
    T *shared_mem_start =  reinterpret_cast<T*>(shared_mem);
    int ptr_bias = 0;
    T* shared_q = GetSharedPtr<T>(shared_mem_start, &ptr_bias, br * head_dim); // size of (br * head_dim)
    T* shared_k = GetSharedPtr<T>(shared_mem_start, &ptr_bias, bc * head_dim); // size of (bc * head_dim)
    T* shared_v = GetSharedPtr<T>(shared_mem_start, &ptr_bias, bc * head_dim); // size of (bc * head_dim)
    T* shared_o = GetSharedPtr<T>(shared_mem_start, &ptr_bias, br * head_dim); // size of (br * head_dim)
    T* shared_l = GetSharedPtr<T>(shared_mem_start, &ptr_bias, br);
    T* shared_l_ij = GetSharedPtr<T>(shared_mem_start, &ptr_bias, br);
    T* shared_l_new = GetSharedPtr<T>(shared_mem_start, &ptr_bias, br);
    T* shared_m = GetSharedPtr<T>(shared_mem_start, &ptr_bias, br);
    T* shared_m_ij = GetSharedPtr<T>(shared_mem_start, &ptr_bias, br);
    T* shared_m_new = GetSharedPtr<T>(shared_mem_start, &ptr_bias, br);
    T* shared_s = GetSharedPtr<T>(shared_mem_start, &ptr_bias, br * bc);
    T* shared_mask = GetSharedPtr<T>(shared_mem_start, &ptr_bias, br * bc);
    for (int j=0;j<outer_steps;++j){
        int row_KV = (j * bc + threadIdx.x);
        // load KV to on-chip memory
        int kv_per_thread = (head_dim + br -1) / br;
        for (int col_idx = 0; col_idx < kv_per_thread; ++col_idx){
            int ele_idx = threadIdx.y * kv_per_thread + col_idx;
            if (row_KV < seq_len && ele_idx < head_dim){
                shared_k[threadIdx.x * head_dim + ele_idx] = K[batch_id * stride_batch + head_id * stride_head + (j * bc + threadIdx.x) * stride_seq + ele_idx];
                shared_v[threadIdx.x * head_dim + ele_idx] = V[batch_id * stride_batch + head_id * stride_head + (j * bc + threadIdx.x) * stride_seq + ele_idx];
            }
        }
        __syncthreads();
        // inner loop
        for (int i = 0;i<inner_steps;++i){
            int row_QO = (i * br + threadIdx.y);
            //load QO to on-chip memory
            int qo_per_thread = (head_dim + bc -1) / bc;
            for (int col_idx = 0; col_idx < qo_per_thread; ++col_idx){
                int ele_idx = threadIdx.x * qo_per_thread + col_idx;
                if (row_QO < seq_len && ele_idx < head_dim){
                    shared_q[threadIdx.y * head_dim + ele_idx] = Q[batch_id * stride_batch + head_id * stride_head + (i * br + threadIdx.y) * stride_seq + ele_idx];
                    shared_o[threadIdx.y * head_dim + ele_idx] = O[batch_id * stride_batch + head_id * stride_head + (i * br + threadIdx.y) * stride_seq + ele_idx];
                }
            }
            // always true for threadIdx.y < bc
            // load l and m to on-chip memory only when j > 0
            if (row_QO < seq_len && threadIdx.x == 0 && j > 0){
                shared_l[threadIdx.y] = L[batch_id * nhead * seq_len + head_id * seq_len + (i * br + threadIdx.y)]; 
                shared_m[threadIdx.y] = M[batch_id * nhead * seq_len + head_id * seq_len + (i * br + threadIdx.y)];
            }
            __syncthreads();
            // compute attention
            if (row_KV < seq_len && row_QO < seq_len){
                T sum_ = 0;
                for (int k = 0; k < head_dim; ++k){
                    sum_ += shared_q[threadIdx.y * head_dim + k] * shared_k[threadIdx.x * head_dim + k];
                }
                shared_s[threadIdx.y * bc + threadIdx.x] = sum_ * rsqrtf(head_dim) ;
            }
            __syncthreads();

            // row-wise computation for the normalization factors
            if (row_QO < seq_len && threadIdx.x == 0){
                T m_ij_ = -FLT_MAX; // need to be -inf
                for (int k = 0; k < bc; ++k){
                   if ((j * bc + k) < seq_len){
                        m_ij_ = fmaxf(m_ij_, shared_s[threadIdx.y * bc + k]);
                   }
                }
                shared_m_ij[threadIdx.y] = m_ij_;
                // softmax
                T l_ij_ = 0;
                for (int k = 0; k < bc; ++k){
                    if ((j * bc + k) < seq_len){
                        shared_s[threadIdx.y * bc + k] = __expf(shared_s[threadIdx.y * bc + k] - m_ij_);
                        l_ij_ += shared_s[threadIdx.y * bc + k];
                    }
                }
                shared_l_ij[threadIdx.y] = l_ij_; //Todo: the shared memory for lij is is actually not needed

                // compute l_new and m_new
                T m_new_, l_new_;
                if (j==0){
                    m_new_ = m_ij_;
                    l_new_ = l_ij_;
                }
                else{
                    m_new_ = fmaxf(shared_m[threadIdx.y], m_ij_);
                    l_new_ = __expf(shared_m[threadIdx.y] - m_new_) * shared_l[threadIdx.y] + __expf(m_ij_ - m_new_) * l_ij_;
                }
                
                // need to write back to shared_memory for row sharing
                shared_l_new[threadIdx.y] = l_new_;
                shared_m_new[threadIdx.y] = m_new_;
            }
            __syncthreads();

            /*
            Compute O on-chip and write back to HBM
            */
            // step 1: compute the shared_o
            T factor_o = shared_l[threadIdx.y] * __expf(shared_m[threadIdx.y] - shared_m_new[threadIdx.y]) / shared_l_new[threadIdx.y];
            T factor_pv = __expf(shared_m_ij[threadIdx.y] - shared_m_new[threadIdx.y]) / shared_l_new[threadIdx.y];
            if (row_QO < seq_len){
                for (int col_idx = 0;col_idx<qo_per_thread; ++col_idx){
                    int ele_idx = threadIdx.x * qo_per_thread + col_idx;
                    if (ele_idx < head_dim){
                        T sum_pv = 0;
                        for (int k = 0; k<bc; ++k){
                            if ((j * bc + k) < seq_len){
                                sum_pv += shared_s[threadIdx.y * bc + k] * shared_v[k * head_dim + ele_idx];
                            }
                        }
                        sum_pv *= factor_pv;
                        if (j==0){
                            shared_o[threadIdx.y * head_dim + ele_idx] = sum_pv; // no previous O and normalization for pv
                        }
                        else{
                            T sum_o = shared_o[threadIdx.y * head_dim + ele_idx] * factor_o + sum_pv;
                            shared_o[threadIdx.y * head_dim + ele_idx] = sum_o;
                        }
                    }
                }
            }
            
            __syncthreads();

            // step 2: write O back to HBM
            if (row_QO < seq_len){
                for (int col_idx = 0; col_idx < qo_per_thread; ++col_idx){
                    int ele_idx = threadIdx.x * qo_per_thread + col_idx;
                    if (ele_idx < head_dim){
                        O[batch_id * stride_batch + head_id * stride_head + (i * br + threadIdx.y) * stride_seq + ele_idx] = shared_o[threadIdx.y * head_dim + ele_idx];
                    }
                }
            }
            // step 3: write l, m back to HBM
            if (row_QO < seq_len && threadIdx.x==0){
                L[batch_id * nhead * seq_len + head_id * seq_len + (i * br + threadIdx.y)] = shared_l_new[threadIdx.y];
                M[batch_id * nhead * seq_len + head_id * seq_len + (i * br + threadIdx.y)] = shared_m_new[threadIdx.y];
            }
            __syncthreads();

        }
    }

}

extern "C" {
void launch_flash_attn_fw(const float *Q, const float* K, const float * V, float * O, 
                                float *L, float *M,
                                int batch_size, int nhead, int seq_len, int head_dim,
                                bool is_causal,
                                hipStream_t stream) {

  int float_size = sizeof(float);
  int qkv_size = batch_size * nhead * seq_len * head_dim * float_size;
  int lm_size = batch_size * nhead * seq_len * float_size;

  float *d_q, *d_k, *d_v, *d_o;
  float *d_l, *d_m;
  hipMalloc((void **)&d_q, qkv_size);
  hipMalloc((void **)&d_k, qkv_size);
  hipMalloc((void **)&d_v, qkv_size);
  hipMalloc((void **)&d_o, qkv_size);
  hipMalloc((void **)&d_l, lm_size);
  hipMalloc((void **)&d_m, lm_size);
//   hipMemset(d_l, 0, lm_size);
//   hipMemset(d_m, 0, lm_size);


  hipMemcpy(d_q, Q, qkv_size, hipMemcpyHostToDevice);
  hipMemcpy(d_k, K, qkv_size, hipMemcpyHostToDevice);
  hipMemcpy(d_v, V, qkv_size, hipMemcpyHostToDevice);


  dim3 grid_dim(nhead, batch_size);

  // get shared memory size M
  hipDeviceProp_t prop;
  int deviceId;
  hipGetDevice(&deviceId);
  hipGetDeviceProperties(&prop, deviceId);
  int Mem = prop.sharedMemPerBlock;

  // calculate block size
  int bc, br;
  bc = min(Mem/(4*head_dim), 16);
  br = min(Mem/(4*head_dim), 16);


  dim3 block_dim(bc, br);

  // launch kernel
  int total_shared_mem_size = ((br * 2 + bc * 2) * head_dim + br * 6 + br * bc) * float_size;
  printf("here get the M size of %d with br size %d, head_dim %d and shared memory size %d\n", Mem, br,head_dim, total_shared_mem_size);

//   flash_attn_fw<float><<<grid_dim, block_dim, total_shared_mem_size, stream>>>(d_q, d_k, d_v, d_o, d_l, d_m, seq_len,head_dim, nullptr, is_causal);
  flash_attn_fw<float><<<grid_dim, block_dim, total_shared_mem_size, stream>>>(d_q, d_k, d_v, d_o, d_l, d_m, nullptr, seq_len,head_dim, nullptr, is_causal);
  
  // Synchronize and check for errors
  hipDeviceSynchronize();
  // Check CUDA execution
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "launch_attn_softmax Error: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  // Copy back to the host
  hipMemcpy(O, d_o, qkv_size, hipMemcpyDeviceToHost);
  hipMemcpy(L, d_l, lm_size, hipMemcpyDeviceToHost);
  hipMemcpy(M, d_m, lm_size, hipMemcpyDeviceToHost);

  // Free memory on device
  hipFree(d_q);
  hipFree(d_k);
  hipFree(d_v);
  hipFree(d_o);
  hipFree(d_l);
  hipFree(d_m);

  
}}


template <typename T>
__global__ void flash_attn_bw(T* dQ, T* dK, T* dV, const T* dO, const T *Q, const T* K, const T* V, const T* O, const T* L, const T* M, int seq_len, int head_dim,const T * masks, bool is_causal) {
    // flash attention bw function
    int batch_id = blockIdx.y;
    int head_id = blockIdx.x;
    int batch_size = gridDim.y;
    int nhead = gridDim.x;
    int br = blockDim.y;
    int bc = blockDim.x;
    int outer_steps = (seq_len + bc - 1) / bc;
    int inner_steps = (seq_len + br - 1) / br;
    
    int stride_batch = nhead * seq_len * head_dim;
    int stride_head = seq_len * head_dim;
    int stride_seq = head_dim;
    // initialize dQ, dK, dV to 0
    // T *dQ = new T[batch_size * nhead * seq_len * head_dim];
    // T *dK = new T[batch_size * nhead * seq_len * head_dim];
    // T *dV = new T[batch_size * nhead * seq_len * head_dim];

    extern __shared__ T shared_mem[];
    T *shared_mem_start =  reinterpret_cast<T*>(shared_mem);
    int ptr_bias = 0;
    T* shared_q = GetSharedPtr<T>(shared_mem_start, &ptr_bias, br * head_dim); // size of (br * head_dim)
    T* shared_k = GetSharedPtr<T>(shared_mem_start, &ptr_bias, bc * head_dim); // size of (bc * head_dim)
    T* shared_v = GetSharedPtr<T>(shared_mem_start, &ptr_bias, bc * head_dim); // size of (bc * head_dim)
    T* shared_o = GetSharedPtr<T>(shared_mem_start, &ptr_bias, br * head_dim); // size of (br * head_dim)
    T* shared_dq = GetSharedPtr<T>(shared_mem_start, &ptr_bias, br * head_dim); // size of (br * head_dim)
    T* shared_dk = GetSharedPtr<T>(shared_mem_start, &ptr_bias, bc * head_dim); // size of (bc * head_dim)
    T* shared_dv = GetSharedPtr<T>(shared_mem_start, &ptr_bias, bc * head_dim); // size of (bc * head_dim)
    T* shared_do = GetSharedPtr<T>(shared_mem_start, &ptr_bias, br * head_dim); // size of (br * head_dim)
    T* shared_l = GetSharedPtr<T>(shared_mem_start, &ptr_bias, br);
    T* shared_m = GetSharedPtr<T>(shared_mem_start, &ptr_bias, br);
    T* shared_s = GetSharedPtr<T>(shared_mem_start, &ptr_bias, br * bc);
    T* shared_p = GetSharedPtr<T>(shared_mem_start, &ptr_bias, br * bc);
    T* shared_ds = GetSharedPtr<T>(shared_mem_start, &ptr_bias, br * bc);
    T* shared_dp = GetSharedPtr<T>(shared_mem_start, &ptr_bias, br * bc);
    T* shared_d = GetSharedPtr<T>(shared_mem_start, &ptr_bias, br);
    // printf("finish initializing shared memory\n");
    for (int j=0;j<outer_steps;++j){
        // load KV to on-chip memory
        // initialize dk, dv = 0 in shared memory
        
        int kv_per_thread = (head_dim + br -1) / br;
        for (int col_idx = 0; col_idx < kv_per_thread; ++col_idx){
            int ele_idx = threadIdx.y * kv_per_thread + col_idx;
            if (ele_idx < head_dim){
                shared_k[threadIdx.x * head_dim + ele_idx] = K[batch_id * stride_batch + head_id * stride_head + (j * bc + threadIdx.x) * stride_seq + ele_idx];
                shared_v[threadIdx.x * head_dim + ele_idx] = V[batch_id * stride_batch + head_id * stride_head + (j * bc + threadIdx.x) * stride_seq + ele_idx];
                // printf("loading kv to on-chip memory\n");
                shared_dk[threadIdx.x * head_dim + ele_idx] = 0;
                // printf("loading dk to on-chip memory\n");
                shared_dv[threadIdx.x * head_dim + ele_idx] = 0;
                // printf("loading dv to on-chip memory\n");
            }
        }
        // printf("finish loading kv to on-chip memory\n");
        __syncthreads();
        // inner loop
        for (int i = 0;i<inner_steps;++i){
            //load Q to on-chip memory

            int qo_per_thread = (head_dim + bc -1) / bc;
            for (int col_idx = 0; col_idx < qo_per_thread; ++col_idx){
                int ele_idx = threadIdx.x * qo_per_thread + col_idx;
                if (ele_idx < head_dim){
                    shared_q[threadIdx.y * head_dim + ele_idx] = Q[batch_id * stride_batch + head_id * stride_head + (i * br + threadIdx.y) * stride_seq + ele_idx];
                    shared_o[threadIdx.y * head_dim + ele_idx] = O[batch_id * stride_batch + head_id * stride_head + (i * br + threadIdx.y) * stride_seq + ele_idx];
                    // printf("loading q to on-chip memory\n");
                    shared_dq[threadIdx.y * head_dim + ele_idx] = dQ[batch_id * stride_batch + head_id * stride_head + (i * br + threadIdx.y) * stride_seq + ele_idx];
                    // printf("loading dq to on-chip memory\n");
                    shared_do[threadIdx.y * head_dim + ele_idx] = dO[batch_id * stride_batch + head_id * stride_head + (i * br + threadIdx.y) * stride_seq + ele_idx];
                    // printf("loading do to on-chip memory\n");
                }
            }
            // printf("finish loading q to on-chip memory\n");
            // always true for threadIdx.y < bc
            // load l and m to on-chip memory
            if (threadIdx.x == 0){
                shared_l[threadIdx.y] = L[batch_id * nhead * seq_len + head_id * seq_len + (i * br + threadIdx.y)]; 
                shared_m[threadIdx.y] = M[batch_id * nhead * seq_len + head_id * seq_len + (i * br + threadIdx.y)];
            }
            // printf("finish loading l and m to on-chip memory\n");
            __syncthreads();
            // compute attention
            T sum_ = 0;
            for (int k = 0; k < head_dim; ++k){
                sum_ += shared_q[threadIdx.y * head_dim + k] * shared_k[threadIdx.x * head_dim + k];
            }
            shared_s[threadIdx.y * bc + threadIdx.x] = sum_ * rsqrtf(head_dim) ;
            // printf("finish computing attention\n");
            __syncthreads();

            // TODO: Add mask

            // calculate p
            shared_p[threadIdx.y * bc + threadIdx.x] = __expf(shared_s[threadIdx.y * bc + threadIdx.x] - shared_m[threadIdx.y]) / shared_l[threadIdx.y];
            // printf("finish computing p\n");
            // TODO: Add dropout
            
            // calcluate dv
            for (int col_idx = 0; col_idx < kv_per_thread; ++col_idx){
                int ele_idx = threadIdx.y * kv_per_thread + col_idx;
                if (ele_idx < head_dim){
                    sum_ = 0;
                    for (int k = 0; k < br; ++k){
                        sum_ += shared_p[k * bc + threadIdx.x] * shared_do[k * head_dim + ele_idx];
                    }
                    shared_dv[threadIdx.x * head_dim + ele_idx] += sum_;
                }
            }
            // printf("finish computing dv\n");
            // calcluate dp
            sum_ = 0;
            for(int k=0;k<head_dim; ++k){
                sum_ += shared_do[threadIdx.y * head_dim + k] * shared_v[threadIdx.x * head_dim + k];
            }
            shared_dp[threadIdx.y * bc + threadIdx.x] = sum_;
            __syncthreads();
            // printf("finish computing dp\n");
            // calculate d, rowsum do*o
            if (threadIdx.x == 0){
                sum_ = 0;
                for (int k = 0; k < head_dim; ++k){
                    sum_ += shared_do[threadIdx.y * head_dim + k] * shared_o[threadIdx.y * head_dim + k];
                }
                shared_d[threadIdx.y] = sum_;
            }
            // printf("finish computing d\n");
            __syncthreads();
            // calculate ds
            shared_ds[threadIdx.y * bc + threadIdx.x] = shared_p[threadIdx.y * bc + threadIdx.x] * (shared_dp[threadIdx.y * bc + threadIdx.x] - shared_d[threadIdx.y]);
            // printf("finish computing ds\n");
            __syncthreads();
            // calculate dq
            for (int col_idx = 0; col_idx < qo_per_thread; ++col_idx){
                int ele_idx = threadIdx.x * qo_per_thread + col_idx;
                if (ele_idx < head_dim){
                    T sum_ = 0;
                    for (int k = 0; k < bc; ++k){
                        sum_ += shared_ds[threadIdx.y * bc + k] * shared_k[k * head_dim + ele_idx];
                    }
                    shared_dq[threadIdx.y * head_dim + ele_idx] += sum_ * rsqrtf(head_dim);
                }
            }
            __syncthreads();
            // printf("finish computing dq\n");
            // write back to HBM dq
            for (int col_idx = 0; col_idx < qo_per_thread; ++col_idx){
                int ele_idx = threadIdx.x * qo_per_thread + col_idx;
                if (ele_idx < head_dim){
                    dQ[batch_id * stride_batch + head_id * stride_head + (i * br + threadIdx.y) * stride_seq + ele_idx] = shared_dq[threadIdx.y * head_dim + ele_idx];
                }
            }
            __syncthreads();
            // printf("finish writing dq back to HBM\n");
            // calculate dk
            for (int col_idx = 0; col_idx < kv_per_thread; ++col_idx){
                int ele_idx = threadIdx.y * kv_per_thread + col_idx;
                if (ele_idx < head_dim){
                    T sum_ = 0;
                    for (int k = 0; k < br; ++k) {
                        sum_ += shared_ds[k * bc + threadIdx.x] * shared_q[k * head_dim + ele_idx];
                    }
                    shared_dk[threadIdx.x * head_dim + ele_idx] += sum_ * rsqrtf(head_dim);
                }
            }
            // printf("finish computing dk\n");
            __syncthreads();
        } // end of inner loop
        // write back to HBM, dv, dk
        for (int col_idx = 0; col_idx < kv_per_thread; ++col_idx){
            int ele_idx = threadIdx.y * kv_per_thread + col_idx;
            if (ele_idx < head_dim){
                dV[batch_id * stride_batch + head_id * stride_head + (j * bc + threadIdx.x) * stride_seq + ele_idx] = shared_dv[threadIdx.x * head_dim + ele_idx];
                dK[batch_id * stride_batch + head_id * stride_head + (j * bc + threadIdx.x) * stride_seq + ele_idx] = shared_dk[threadIdx.x * head_dim + ele_idx];
            }
        }
        __syncthreads(); 
    } // end of outer loop
} // flash_attn_bw

extern "C" {
void launch_flash_attn_bw(float *dQ, float *dK, float *dV, const float *dO, const float *Q, const float* K, const float * V, const float * O,
                                const float *L, const float *M,
                                int batch_size, int nhead, int seq_len, int head_dim,
                                bool is_causal,
                                hipStream_t stream) {

    int float_size = sizeof(float);
    int qkv_size = batch_size * nhead * seq_len * head_dim * float_size;
    int lm_size = batch_size * nhead * seq_len * float_size;

    float *d_q, *d_k, *d_v, *d_o;
    float *grad_q, *grad_k, *grad_v, *grad_o;
    float * d_l, *d_m;
    hipMalloc((void **)&d_q, qkv_size);
    hipMalloc((void **)&d_k, qkv_size);
    hipMalloc((void **)&d_v, qkv_size);
    hipMalloc((void **)&d_o, qkv_size);
    hipMalloc((void **)&d_l, lm_size);
    hipMalloc((void **)&d_m, lm_size);
    hipMalloc((void **)&grad_q, qkv_size);
    hipMalloc((void **)&grad_k, qkv_size);
    hipMalloc((void **)&grad_v, qkv_size);
    hipMalloc((void **)&grad_o, qkv_size);
    // hipMemset(d_l, 0, lm_size);
    // hipMemset(d_m, 0, lm_size);

    hipMemcpy(d_q, Q, qkv_size, hipMemcpyHostToDevice);
    hipMemcpy(d_k, K, qkv_size, hipMemcpyHostToDevice);
    hipMemcpy(d_v, V, qkv_size, hipMemcpyHostToDevice);
    hipMemcpy(d_o, O, qkv_size, hipMemcpyHostToDevice);
    hipMemcpy(d_l, L, lm_size, hipMemcpyHostToDevice);
    hipMemcpy(d_m, M, lm_size, hipMemcpyHostToDevice);
    hipMemcpy(grad_o, dO, qkv_size, hipMemcpyHostToDevice);

    dim3 grid_dim(nhead, batch_size);

    // get shared memory size M
    hipDeviceProp_t prop;
    int deviceId;
    hipGetDevice(&deviceId);
    hipGetDeviceProperties(&prop, deviceId);
    int Mem = prop.sharedMemPerBlock;

    // calculate block size
    int bc, br;
    bc = min(Mem/(4*head_dim), 16);
    br = min(Mem/(4*head_dim), 16);

    dim3 block_dim(bc, br);

    // launch kernel
    int total_shared_mem_size = ((br * 4 + bc * 4) * head_dim + br * 3 + 4 * (br * bc)) * float_size;
    printf("here get the M size of %d with br size %d, head_dim %d and shared memory size %d\n", Mem, br,head_dim, total_shared_mem_size);

    flash_attn_bw<float><<<grid_dim, block_dim, total_shared_mem_size, stream>>>(grad_q, grad_k, grad_v, grad_o, d_q, d_k, d_v, d_o, d_l, d_m, seq_len,head_dim, nullptr, is_causal);
    // Synchronize and check for errors
    hipDeviceSynchronize();
    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "launch_attn_softmax_bw Error: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Copy back to the host
    hipMemcpy(dQ, grad_q, qkv_size, hipMemcpyDeviceToHost);
    hipMemcpy(dK, grad_k, qkv_size, hipMemcpyDeviceToHost);
    hipMemcpy(dV, grad_v, qkv_size, hipMemcpyDeviceToHost);

    // Free memory on device
    hipFree(d_q);
    hipFree(d_k);
    hipFree(d_v);
    hipFree(d_o);
    hipFree(d_l);
    hipFree(d_m);
    hipFree(grad_q);
    hipFree(grad_k);
    hipFree(grad_v);
    hipFree(grad_o);

} // launch_attn_softmax_bw
} // extern "C"
    
    } // namespace cuda
    } // namespace flash_attn
