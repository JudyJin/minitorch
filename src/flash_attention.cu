#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>

#include <cub/block/block_load.cuh>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

#include "includes/block_reduce.h"
#include "includes/kernels.h"

#include <hip/hip_cooperative_groups.h>

namespace flash_attn{
namespace cuda{

// called from global, use pointer to modify the bias in-place
template <typename T>
__device__ T* GetSharedPtr(T * shared_mem, int * ptr_bias, int mem_size) {
    T * cur_shared_mem = shared_mem + *ptr_bias;
    *ptr_bias += mem_size;
    return cur_shared_mem;
}
// remember that br/bc might divide seq_len


template <typename T>
__global__ void flash_attn_fw(const T *Q, const T* K, const T* V, T* O, T* L, T* M, int seq_len, int head_dim,const T * masks, bool is_causal) {
    int batch_id = blockIdx.y;
    int head_id = blockIdx.z;
    int batch_size = gridDim.y;
    int nhead = gridDim.z;
    int br = blockDim.y;
    int bc = blockDim.x;
    int outer_steps = (seq_len + bc - 1) / bc;
    int inner_steps = (seq_len + br - 1) / br;

    int stride_batch = nhead * seq_len * head_dim;
    int stride_head = seq_len * head_dim;
    int stride_seq = head_dim;
    // typedef hipcub::BlockLoad<T, block_dim, ele_per_thread,
    //                      hipcub::BLOCK_LOAD_VECTORIZE>
    //     BlockLoad;
    // __shared__ typename BlockLoad::TempStorage ts_load;
    // typedef hipcub::BlockStore<T, block_dim, ele_per_thread,
    //                         hipcub::BLOCK_STORE_VECTORIZE>
    //     BlockStore;
    // __shared__ typename BlockStore::TempStorage ts_store;

    extern __shared__ T shared_mem[];
    T *shared_mem_start =  reinterpret_cast<T*>(shared_mem);
    int ptr_bias = 0;
    T* shared_q = GetSharedPtr<T*>(shared_mem_start, &ptr_bias, br * head_dim); // size of (br * head_dim)
    T* shared_k = GetSharedPtr<T*>(shared_mem_start, &ptr_bias, bc * head_dim); // size of (bc * head_dim)
    T* shared_v = GetSharedPtr<T*>(shared_mem_start, &ptr_bias, bc * head_dim); // size of (bc * head_dim)
    T* shared_o = GetSharedPtr<T*>(shared_mem_start, &ptr_bias, br * head_dim); // size of (br * head_dim)
    T* shared_l = GetSharedPtr<T*>(shared_mem_start, &ptr_bias, br);
    T* shared_l_ij = GetSharedPtr<T*>(shared_mem_start, &ptr_bias, br);
    T* shared_l_new = GetSharedPtr<T*>(shared_mem_start, &ptr_bias, br);
    T* shared_m = GetSharedPtr<T*>(shared_mem_start, &ptr_bias, br);
    T* shared_m_ij = GetSharedPtr<T*>(shared_mem_start, &ptr_bias, br);
    T* shared_m_new = GetSharedPtr<T*>(shared_mem_start, &ptr_bias, br);
    T* shared_s = GetSharedPtr<T*>(shared_mem_start, &ptr_bias, br * bc);
    
    // __shared__ T shared_q[br][head_dim];
    // __shared__ T shared_o[br][bc];
    for (int j=0;j<outer_steps;++j){
        // load KV to on-chip memory
        int kv_per_thread = (head_dim + br -1) / br;
        for (int col_idx = 0; col_idx < kv_per_thread; ++col_idx){
            int ele_idx = threadIdx.y * num_per_thread + col_idx;
            if (ele_idx < head_dim){
                shared_k[threadIdx.x * head_dim + ele_idx] = K[batch_id * stride_batch + head_id * stride_head + (j * bc + threadIdx.x) * stride_seq + ele_idx];
                shared_v[threadIdx.x * head_dim + ele_idx] = V[batch_id * stride_batch + head_id * stride_head + (j * bc + threadIdx.x) * stride_seq + ele_idx];
            }
        }
        __syncthreads();
        // inner loop
        for (int i = 0;i<inner_steps;++i){
            //load Q to on-chip memory
            int qo_per_thread = (head_dim + br -1) / br;
            for (int col_idx = 0; col_idx < qo_per_thread; ++col_idx){
                int ele_idx = threadIdx.x * num_per_thread + col_idx;
                if (ele_idx < head_dim){
                    shared_q[threadIdx.y * head_dim + ele_idx] = Q[batch_id * stride_batch + head_id * stride_head + (i * br + threadIdx.y) * stride_seq + ele_idx];
                }
            }
            // always true for threadIdx.y < bc
            // load l and m to on-chip memory
            if (threadIdx.x == 0){
                shared_l[threadIdx.y] = L[batch_id * n_head * seq_len + head_id * seq_len + (i * br + threadIdx.y)]; 
                shared_m[threadIdx.y] = M[batch_id * n_head * seq_len + head_id * seq_len + (i * br + threadIdx.y)];
            }
            __syncthreads();
            // compute attention
            T sum_ = 0;
            for (int k = 0; k < head_dim; ++k){
                sum_ += shared_q[threadIdx.y * head_dim + k] * shared_k[threadIdx.x * head_dim + k];
            }
            shared_s[threadIdx.y * bc + threadIdx.x] = sum_;
            
            __syncthreads();

            // row-wise computation for the normalization factors
            if (threadIdx.x == 0){
                T m_ij_ = 0;
                for (int k = 0; k < bc; ++k){
                    m_ij_ = max(m_ij_, shared_s[threadIdx.y][k]);
                }
                shared_m_ij[threadIdx.y] = mij_;
                // softmax
                T l_ij_ = 0;
                for (int k = 0; k < bc; ++k){
                    shared_s[threadIdx.y * bc + k] = __expf(shared_s[threadIdx.y * bc + k] - m_ij_);
                    l_ij_ += shared_s[threadIdx.y * bc + k];
                }
                shared_l_ij[threadIdx.y] = l_ij_;

                // compute l_new and m_new
                T m_new_ = max(shared_m[threadIdx.y], m_ij_);
                T l_new_ = __expf(shared_m[threadIdx.y] - m_new_) * shared_l[threadIdx.y] + __expf(m_ij_ - m_new_) * l_ij_;
                
                // need to write back to shared_memory for row sharing
                shared_l_new[threadIdx.y] = l_new_;
                shared_m_new[threadIdx.y] = m_new_;
            }
            __syncthreads();

            /*
            Compute O on-chip and write back to HBM
            */
            // step 1: compute the shared_o
            T factor_o = shared_l[threadIdx.y] * __expf(shared_m[threadIdx.y] - shared_m_new[threadIdx.y]) / shared_l_new[threadIdx.y];
            T factor_pv = __expf(shared_m_ij[threadIdx.y] - shared_m_new[threadIdx.y]) / shared_l_new[threadIdx.y];
            for (int col_idx = 0;col_idx<qo_per_thread; ++col_idx){
                int ele_idx = threadIdx.x * num_per_thread + col_idx;
                if (ele_idx < head_dim){
                    T sum_pv = 0;
                    for (int k = 0; k<bc; ++k){
                        sum_pv += shared_s[threadIdx.y * bc + k] * shared_v[k * head_dim + ele_idx];
                    }
                    sum_pv *= factor_pv;
                    T sum_o = shared_o[threadIdx.y * head_dim + ele_idx] * factor_o + sum_pv;
                    shared_o[threadIdx.y * head_dim + ele_idx] = sum_o;
                }
            }
            __syncthreads();

            // step 2: write O back to HBM
            for (int col_idx = 0; col_idx < qo_per_thread; ++col_idx){
                int ele_idx = threadIdx.x * num_per_thread + col_idx;
                if (ele_idx < head_dim){
                    O[batch_id * stride_batch + head_id * stride_head * (i * br + threadIdx.y) * stride_seq + ele_idx] = shared_o[threadIdx.y * head_dim + ele_idx];
                }
            }
            // step 3: write l, m back to HBM
            if (threadIdx.x==0){
                L[batch_id * n_head * seq_len + head_id * seq_len + (i * br + threadIdx.y)] = shared_l_new[threadIdx.y];
                M[batch_id * n_head * seq_len + head_id * seq_len + (i * br + threadIdx.y)] = shared_m_new[threadIdx.y];
            }
            __syncthreads();

        }

    }

}
    

extern "C" {
void launch_flash_attn_fw(const float *Q, const float* K, const float * V, float * O, 
                                int batch_size, int nhead, int seq_len, int head_dim,
                                bool is_causal,
                                hipStream_t stream) {

  int float_size = sizeof(float);
  int qkv_size = batch_size * nhead * seq_len * head_dim * float_size;

  float *d_q, *d_k, *d_v, *d_o;
  hipMalloc((void **)&d_q, qkv_size);
  hipMalloc((void **)&d_k, qkv_size);
  hipMalloc((void **)&d_v, qkv_size);
  hipMalloc((void **)&d_o, qkv_size);


  hipMemcpy(d_q, Q, qkv_size, hipMemcpyHostToDevice);
  hipMemcpy(d_k, K, qkv_size, hipMemcpyHostToDevice);
  hipMemcpy(d_v, V, qkv_size, hipMemcpyHostToDevice);


  dim3 grid_dim(1, batch_size, nhead);

  // get shared memory size M
  hipDeviceProp_t prop;
  int deviceId;
  hipGetDevice(&deviceId);
  hipGetDeviceProperties(&prop, deviceId);
  int M = prop.sharedMemPerBlock;

  // calculate block size
  int bc, br;
  bc = min(M/(4*head_dim), 32);
  br = min(M/(4*head_dim), 32);

  dim3 block_dim(bc, br);

  // launch kernel
  int total_shared_mem_size = ((br * 2 + bc * 2) * head_dim + br * 6 + br * bc) * float_size;
  flash_attn_fw<float><<<grid_dim, block_dim, total_shared_mem_size, stream>>>(d_q, d_k, d_v, d_o, head_dim, nullptr, is_causal);
  
  // Copy back to the host
  hipMemcpy(O, d_o, qkv_size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  // Check CUDA execution
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "launch_attn_softmax Error: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Free memory on device
  hipFree(d_q);
  hipFree(d_k);
  hipFree(d_v);
  hipFree(d_o);
  
}}


    
}
}